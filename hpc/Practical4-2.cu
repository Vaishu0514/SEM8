#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void matmul(int* A, int* B, int* C) {
    int row = threadIdx.x;
    int col = threadIdx.y;

    if (row < 2 && col < 2) {
        int sum = 0;
        for (int k = 0; k < 2; k++) {
            sum += A[row * 2 + k] * B[k * 2 + col];
        }
        C[row * 2 + col] = sum;
    }
}

int main() {
    // Declare matrices A, B, and C
    int A[2][2] = {{1, 2}, {3, 4}};  // Example 2x2 matrix A
    int B[2][2] = {{5, 6}, {7, 8}};  // Example 2x2 matrix B
    int C[2][2];  // Result matrix C

    int *d_A, *d_B, *d_C;
    int size = 2 * 2 * sizeof(int); // Size of 2x2 matrix

    // Allocate memory on the device (GPU)
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy data from host to device
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    // Launch the kernel with a 2x2 block size (one thread per element)
    dim3 threadsPerBlock(2, 2); // 2x2 block of threads
    matmul<<<1, threadsPerBlock>>>(d_A, d_B, d_C);

    // Check for any errors during kernel launch
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
        return -1;
    }

    // Copy the result back from device to host
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    // Print the result matrix C
    printf("Resulting matrix C:\n");
    for (int i = 0; i < 2; i++) {
        for (int j = 0; j < 2; j++) {
            printf("%d ", C[i][j]);
        }
        printf("\n");
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
