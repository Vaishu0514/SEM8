#include <iostream>
#include <vector>
#include <queue>
#include <unordered_map>
#include <hip/hip_runtime.h>

// Huffman Node structure
struct HuffmanNode {
    unsigned char data;
    unsigned int frequency;
    HuffmanNode* left;
    HuffmanNode* right;

    HuffmanNode(unsigned char data, unsigned int frequency) : data(data), frequency(frequency), left(nullptr), right(nullptr) {}
};

// Comparison function for priority queue
struct CompareNodes {
    bool operator()(HuffmanNode* left, HuffmanNode* right) {
        return left->frequency > right->frequency;
    }
};

// Function to build Huffman tree (CPU)
HuffmanNode* buildHuffmanTree(const std::unordered_map<unsigned char, unsigned int>& frequencies) {
    std::priority_queue<HuffmanNode*, std::vector<HuffmanNode*>, CompareNodes> minHeap;

    for (const auto& pair : frequencies) {
        minHeap.push(new HuffmanNode(pair.first, pair.second));
    }

    while (minHeap.size() > 1) {
        HuffmanNode* left = minHeap.top();
        minHeap.pop();
        HuffmanNode* right = minHeap.top();
        minHeap.pop();

        HuffmanNode* parent = new HuffmanNode('\0', left->frequency + right->frequency);
        parent->left = left;
        parent->right = right;

        minHeap.push(parent);
    }

    return minHeap.top();
}

// Function to generate Huffman codes (CPU)
void generateHuffmanCodes(HuffmanNode* root, std::unordered_map<unsigned char, std::string>& huffmanCodes, std::string code = "") {
    if (root == nullptr) {
        return;
    }

    if (root->data != '\0') {
        huffmanCodes[root->data] = code;
    }

    generateHuffmanCodes(root->left, huffmanCodes, code + "0");
    generateHuffmanCodes(root->right, huffmanCodes, code + "1");
}

// CUDA kernel for frequency counting
__global__ void countFrequencies(const unsigned char* input, int inputSize, unsigned int* frequencies) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < inputSize) {
        atomicAdd(&frequencies[input[tid]], 1);
    }
}

// CUDA kernel for encoding
__global__ void encodeData(const unsigned char* input, int inputSize, const unsigned int* huffmanCodeLengths, const unsigned int* huffmanCodeValues, unsigned char* output, int* outputSize) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < inputSize) {
        unsigned char symbol = input[tid];
        unsigned int codeLength = huffmanCodeLengths[symbol];
        unsigned int codeValue = huffmanCodeValues[symbol];
        // In a real implementation, you'd pack the bits into the output buffer. This example skips that for simplicity.
        // This example only prints the code, not the actual encoding.
        if (threadIdx.x == 0) printf("Symbol: %c, Code: %u, Length: %u\n", symbol, codeValue, codeLength);
    }
}

int main() {
    std::string inputString = "Bubblegum";
    int inputSize = inputString.size();
    const unsigned char* input = reinterpret_cast<const unsigned char*>(inputString.c_str());

    // Allocate memory on the host
    unsigned int* hostFrequencies = new unsigned int[256]();
    unsigned char* hostOutput = new unsigned char[inputSize * 8]; // Assuming max 8 bits per character
    int hostOutputSize = 0;

    // Allocate memory on the device
    unsigned char* deviceInput;
    unsigned int* deviceFrequencies;
    unsigned char* deviceOutput;
    int* deviceOutputSize;

    hipMalloc(&deviceInput, inputSize);
    hipMalloc(&deviceFrequencies, 256 * sizeof(unsigned int));
    hipMalloc(&deviceOutput, inputSize * 8);
    hipMalloc(&deviceOutputSize, sizeof(int));

    // Copy input data to the device
    hipMemcpy(deviceInput, input, inputSize, hipMemcpyHostToDevice);
    hipMemset(deviceFrequencies, 0, 256 * sizeof(unsigned int)); // Initialize frequencies to 0

    // Launch frequency counting kernel
    int blockSize = 256;
    int gridSize = (inputSize + blockSize - 1) / blockSize;
    countFrequencies<<<gridSize, blockSize>>>(deviceInput, inputSize, deviceFrequencies);

    // Copy frequencies back to the host
    hipMemcpy(hostFrequencies, deviceFrequencies, 256 * sizeof(unsigned int), hipMemcpyDeviceToHost);

    // Build frequency map for CPU tree building
    std::unordered_map<unsigned char, unsigned int> frequencies;
    for (int i = 0; i < 256; ++i) {
        if (hostFrequencies[i] > 0) {
            frequencies[static_cast<unsigned char>(i)] = hostFrequencies[i];
        }
    }

    // Build Huffman tree and generate codes (CPU)
    HuffmanNode* root = buildHuffmanTree(frequencies);
    std::unordered_map<unsigned char, std::string> huffmanCodes;
    generateHuffmanCodes(root, huffmanCodes);

    // Prepare code lengths and values for GPU encoding
    unsigned int hostCodeLengths[256] = { 0 };
    unsigned int hostCodeValues[256] = { 0 };
    for (const auto& pair : huffmanCodes) {
        hostCodeLengths[pair.first] = pair.second.length();
        hostCodeValues[pair.first] = std::stoul(pair.second, nullptr, 2); // Convert binary string to int
    }

    unsigned int* deviceCodeLengths;
    unsigned int* deviceCodeValues;

    hipMalloc(&deviceCodeLengths, 256 * sizeof(unsigned int));
    hipMalloc(&deviceCodeValues, 256 * sizeof(unsigned int));

    hipMemcpy(deviceCodeLengths, hostCodeLengths, 256 * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(deviceCodeValues, hostCodeValues, 256 * sizeof(unsigned int), hipMemcpyHostToDevice);

    // Launch encoding kernel (example output)
    encodeData<<<gridSize, blockSize>>>(deviceInput, inputSize, deviceCodeLengths, deviceCodeValues, deviceOutput, deviceOutputSize);

    // Clean up
    hipFree(deviceInput);
    hipFree(deviceFrequencies);
    hipFree(deviceOutput);
    hipFree(deviceOutputSize);
    hipFree(deviceCodeLengths);
    hipFree(deviceCodeValues);
    delete[] hostFrequencies;
    delete[] hostOutput;

    return 0;
}
